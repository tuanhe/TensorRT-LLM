#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION &
 * AFFILIATES. All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/runtime/runtimeKernels.h"

#include <hipcub/hipcub.hpp>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

using namespace tensorrt_llm::runtime;
namespace tc = tensorrt_llm::common;

namespace tensorrt_llm::runtime::kernels
{

namespace
{

template <typename T>
__global__ void fill(T* data, std::size_t size, T const value)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] = value;
    }
}
} // namespace

template <typename T>
void invokeFill(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    fill<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

// template instantiation
template void invokeFill(IBuffer&, std::int32_t, CudaStream const&);
template void invokeFill(IBuffer&, std::int8_t, CudaStream const&);
template void invokeFill(IBuffer&, float, CudaStream const&);

namespace
{
template <typename T>
__global__ void fillBatch(T* data, std::int32_t const* indices, std::size_t size, T const* values)
{
    auto const batchIdx = indices[blockIdx.y];
    const T value = values[blockIdx.y];
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const startIdx = batchIdx * size;
    auto const endIdx = startIdx + size;

    for (auto idx = startIdx + tidx; idx < endIdx; idx += stride)
    {
        data[idx] = value;
    }
}
} // namespace

template <typename T>
void invokeFillBatch(IBuffer& buffer, IBuffer const& slotIndices, std::size_t slotStride, IBuffer const& values,
    CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto indices = bufferCast<std::int32_t>(slotIndices);
    auto fillValues = bufferCast<T>(values);
    auto numSlots = slotIndices.getSize();
    auto const size = slotStride;
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};

    fillBatch<<<gridSize, blockSize, 0, stream.get()>>>(data, indices, size, fillValues);
}

// template instantiation
template void invokeFillBatch<float>(IBuffer&, IBuffer const&, std::size_t, IBuffer const&, CudaStream const&);
template void invokeFillBatch<std::int8_t>(IBuffer&, IBuffer const&, std::size_t, IBuffer const&, CudaStream const&);
template void invokeFillBatch<std::int32_t>(IBuffer&, IBuffer const&, std::size_t, IBuffer const&, CudaStream const&);

namespace
{
template <typename T>
__global__ void copyBatch(
    const T* srcData, T* dstData, std::int32_t const* srcIndices, std::int32_t const* dstIndices, std::size_t size)
{
    auto const srcBatchIdx = srcIndices[blockIdx.y];
    auto const dstBatchIdx = dstIndices[blockIdx.y];
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const srcStartIdx = srcBatchIdx * size;
    auto const dstStartIdx = dstBatchIdx * size;
    auto const srcEndIdx = srcStartIdx + size;

    auto srcIdx = srcStartIdx + tidx;
    auto dstIdx = dstStartIdx + tidx;

    for (; srcIdx < srcEndIdx; srcIdx += stride, dstIdx += stride)
    {
        dstData[dstIdx] = srcData[srcIdx];
    }
}
} // namespace

template <typename T>
void invokeCopyBatch(IBuffer const& srcBuffer, IBuffer& dstBuffer, IBuffer const& srcIndices, IBuffer const& dstIndices,
    std::size_t slotStride, CudaStream const& stream)
{
    auto srcDataPtr = bufferCast<T>(srcBuffer);
    auto dstDataPtr = bufferCast<T>(dstBuffer);
    auto srcIndicesPtr = bufferCast<std::int32_t>(srcIndices);
    auto dstIndicesPtr = bufferCast<std::int32_t>(dstIndices);
    auto numSlots = srcIndices.getSize();
    auto const size = slotStride;
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};

    copyBatch<<<gridSize, blockSize, 0, stream.get()>>>(srcDataPtr, dstDataPtr, srcIndicesPtr, dstIndicesPtr, size);
}

// template instantiation
template void invokeCopyBatch<float>(
    IBuffer const&, IBuffer&, IBuffer const&, IBuffer const&, std::size_t, CudaStream const&);
template void invokeCopyBatch<std::int8_t>(
    IBuffer const&, IBuffer&, IBuffer const&, IBuffer const&, std::size_t, CudaStream const&);
template void invokeCopyBatch<std::int32_t>(
    IBuffer const&, IBuffer&, IBuffer const&, IBuffer const&, std::size_t, CudaStream const&);

namespace
{
template <typename T>
__global__ void add(T* data, std::size_t size, T const value)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] += value;
    }
}
} // namespace

template <typename T>
void invokeAdd(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    add<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

template void invokeAdd(IBuffer&, std::int32_t, CudaStream const&);
template void invokeAdd(IBuffer&, std::int8_t, CudaStream const&);
template void invokeAdd(IBuffer&, float, CudaStream const&);

namespace
{
template <typename T>
__global__ void reduceSum(T* output, T const* input, std::size_t size)
{
    T threadSum = 0;
    for (auto index = threadIdx.x; index < size; index += blockDim.x)
    {
        threadSum += input[index];
    }

    T blockSum = 0;
    if (blockDim.x <= 32)
    {
        blockSum = tc::warpReduceSum(threadSum);
    }
    else
    {
        blockSum = tc::blockReduceSum(threadSum);
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        *output = blockSum;
    }
}
} // namespace

template <typename T>
void invokeReduce(IBuffer& output, IBuffer const& input, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");
    TLLM_CHECK_WITH_INFO(output.getSize() == 1, common::fmtstr("Output size (%ld) has to be 1", output.getSize()));

    auto outputPtr = bufferCast<T>(output);
    auto inputPtr = bufferCast<T>(input);
    auto const size = input.getSize();

    dim3 blockSize{std::min(512u, static_cast<std::uint32_t>(size))};
    dim3 gridSize{1};

    reduceSum<<<gridSize, blockSize, 0, stream.get()>>>(outputPtr, inputPtr, size);
}

void reduce(IBuffer& output, IBuffer const& input, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeReduce<SizeType>(output, input, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeReduce<float>(output, input, stream); break;
    case nvinfer1::DataType::kHALF: invokeReduce<half>(output, input, stream); break;
    case nvinfer1::DataType::kINT8: invokeReduce<int8_t>(output, input, stream); break;
    default: TLLM_CHECK_WITH_INFO(false, "data type not supported");
    }
}

namespace
{
__global__ void transpose(SizeType* output, SizeType const* input, SizeType const batchSize, SizeType const rowSize)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType tokenIdx = tidx; tokenIdx < rowSize; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * rowSize + tokenIdx;
            auto const outputIdx = tokenIdx * batchSize + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTranspose(ITensor& output, ITensor const& input, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");
    TLLM_CHECK_WITH_INFO(input.getSize() == output.getSize(),
        common::fmtstr("Input size (%ld) and output size (%ld) differ", input.getSize(), output.getSize()));

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));

    SizeType const batchSize = inputShape.d[0];
    SizeType const rowSize = inputShape.d[1];

    dim3 const blockSize(256, 1);
    dim3 const gridSize((rowSize + blockSize.x - 1) / blockSize.x, batchSize);

    transpose<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<SizeType>(output), bufferCast<SizeType const>(input), batchSize, rowSize);
}

namespace
{
__global__ void transposeWithOutputOffset(SizeType* output, SizeType const* input, SizeType const nbInputRows,
    SizeType const inputRowSize, SizeType const outputRowSize, SizeType const outputOffset)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < nbInputRows; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType tokenIdx = tidx; tokenIdx < inputRowSize; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * inputRowSize + tokenIdx;
            auto const outputIdx = tokenIdx * outputRowSize + outputOffset + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTransposeWithOutputOffset(
    ITensor& output, ITensor const& input, SizeType const outputOffset, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));
    SizeType const nbInputRows = inputShape.d[0];
    SizeType const inputRowSize = inputShape.d[1];

    auto const& outputShape = output.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 2, common::fmtstr("Output shape must have 2 dimensions, but has %d", outputShape.nbDims));
    SizeType const nbOutputRows = outputShape.d[0];
    SizeType const outputRowSize = outputShape.d[1];

    TLLM_CHECK_WITH_INFO(inputRowSize == nbOutputRows,
        common::fmtstr("Input dim 1 (%d) and output dim 0 (%d) differ", inputRowSize, nbOutputRows));
    TLLM_CHECK_WITH_INFO(outputOffset + nbInputRows <= outputRowSize,
        common::fmtstr("Input (%d rows) does not fit into output (%d columns, offset %d)", nbInputRows, inputRowSize,
            outputOffset));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((inputRowSize + blockSize.x - 1) / blockSize.x, nbInputRows);

    transposeWithOutputOffset<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(output),
        bufferCast<SizeType const>(input), nbInputRows, inputRowSize, outputRowSize, outputOffset);
}

namespace
{
__global__ void transposeWithInputOffset(SizeType* output, SizeType const* input, SizeType const outputRowSize,
    SizeType const nbOutputRows, SizeType const inputRowSize, SizeType const inputOffset)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < outputRowSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType tokenIdx = tidx; tokenIdx < nbOutputRows; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * inputRowSize + inputOffset + tokenIdx;
            auto const outputIdx = tokenIdx * outputRowSize + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTransposeWithInputOffset(
    ITensor& output, ITensor const& input, SizeType const inputOffset, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));
    SizeType const nbInputRows = inputShape.d[0];
    SizeType const inputRowSize = inputShape.d[1];

    auto const& outputShape = output.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 2, common::fmtstr("Output shape must have 2 dimensions, but has %d", outputShape.nbDims));
    SizeType const nbOutputRows = outputShape.d[0];
    SizeType const outputRowSize = outputShape.d[1];

    TLLM_CHECK_WITH_INFO(nbInputRows == outputRowSize,
        common::fmtstr("Input dim 0 (%d) and output dim 1 (%d) differ", nbInputRows, outputRowSize));
    TLLM_CHECK_WITH_INFO(inputOffset + nbOutputRows <= inputRowSize,
        common::fmtstr("Cannot extract output (%d rows) from input (%d columns, offset %d)", nbOutputRows, inputRowSize,
            inputOffset));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((nbOutputRows + blockSize.x - 1) / blockSize.x, outputRowSize);

    transposeWithInputOffset<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(output),
        bufferCast<SizeType const>(input), outputRowSize, nbOutputRows, inputRowSize, inputOffset);
}

void invokeInclusiveSum(IBuffer& output, IBuffer const& input, BufferManager const& manager, CudaStream const& stream)
{
    auto const size = input.getSize();
    auto const* inputData = bufferCast<SizeType>(input);
    auto* outputData = bufferCast<SizeType>(output);

    std::size_t tempStorageBytes{0};
    hipcub::DeviceScan::InclusiveSum(nullptr, tempStorageBytes, inputData, outputData, size, stream.get());
    auto tempStorage = manager.gpu(tempStorageBytes, nvinfer1::DataType::kUINT8);
    auto* tempStorageData = bufferCast<std::uint8_t>(*tempStorage);
    hipcub::DeviceScan::InclusiveSum(tempStorageData, tempStorageBytes, inputData, outputData, size, stream.get());
}

namespace
{
__global__ void buildTokenMask(SizeType* tokenMask, SizeType const* inputLengths, SizeType const batchSize,
    SizeType const maxInputLength, SizeType const maxSeqLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType tokenIdx = tidx; tokenIdx < maxSeqLength; tokenIdx += blockDim.x * gridDim.x)
        {
            tokenMask[batchIdx * maxSeqLength + tokenIdx]
                = (tokenIdx >= inputLength && tokenIdx < maxInputLength) ? 1 : 0;
        }
    }
}
} // namespace

void invokeBuildTokenMask(
    ITensor& tokenMask, ITensor const& inputLengths, SizeType const maxInputLength, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType>::value == tokenMask.getDataType(), "tokenMask has wrong data type");
    TLLM_CHECK_WITH_INFO(
        TRTDataType<SizeType>::value == inputLengths.getDataType(), "inputLengths has wrong data type");

    auto const& shape = tokenMask.getShape();
    SizeType const batchSize = shape.d[0];
    SizeType const maxSeqLength = shape.d[1];

    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "TtokenMask dimension 1 (%d) is smaller than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxSeqLength + blockSize.x - 1) / blockSize.x, batchSize);

    buildTokenMask<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(tokenMask),
        bufferCast<SizeType const>(inputLengths), batchSize, maxInputLength, maxSeqLength);
}

namespace
{
__global__ void buildAttentionMask(SizeType* attentionMask, SizeType const size, SizeType const padId)
{
    SizeType const tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (SizeType i = tid; i < size; i += blockDim.x * gridDim.x)
    {
        auto const x = attentionMask[i];
        attentionMask[i] = (x != padId);
    }
}
} // namespace

void invokeBuildAttentionMask(ITensor& attentionMask, SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        TRTDataType<SizeType>::value == attentionMask.getDataType(), "attentionMask has wrong data type");

    auto const size = attentionMask.getSize();
    dim3 const blockSize(256);
    dim3 const gridSize((size + blockSize.x - 1) / blockSize.x);

    buildAttentionMask<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(attentionMask), size, padId);
}

namespace
{
__global__ void extendAttentionMask(
    SizeType* newMask, SizeType const* oldMask, SizeType const batchSize, SizeType const seqLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType tokenIdx = tidx; tokenIdx < seqLength + 1; tokenIdx += blockDim.x * gridDim.x)
        {
            SizeType oldIndex = batchIdx * seqLength + tokenIdx;
            SizeType newIndex = batchIdx * (seqLength + 1) + tokenIdx;
            newMask[newIndex] = (tokenIdx < seqLength) ? oldMask[oldIndex] : 1;
        }
    }
}
} // namespace

void invokeExtendAttentionMask(ITensor& newMask, ITensor const& oldMask, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType>::value == newMask.getDataType(), "attentionMask has wrong data type");
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType>::value == oldMask.getDataType(), "attentionMask has wrong data type");

    auto const& shape = oldMask.getShape();
    SizeType const batchSize = shape.d[0];
    SizeType const seqLength = shape.d[1];

    dim3 const blockSize(256, 1);
    dim3 const gridSize((seqLength + blockSize.x - 1) / blockSize.x, batchSize);

    extendAttentionMask<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<SizeType>(newMask), bufferCast<SizeType>(oldMask), batchSize, seqLength);
}

namespace
{
__global__ void copyInputToOutputTransposed(SizeType* outputIds, SizeType const* inputIds, SizeType const* inputLengths,
    SizeType const padId, SizeType const batchSize, SizeType const beamWidth, SizeType const maxInputLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[batchIdx * maxInputLength + tokenIdx] : padId;
            for (SizeType beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(tokenIdx, batchIdx, beamIdx, batchSize, beamWidth);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyInputToOutputTransposed(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const batchSize = static_cast<SizeType>(inputLengths.getSize());
    auto const& inputShape = inputIds.getShape();
    SizeType const maxInputLength = inputShape.d[inputShape.nbDims - 1];
    auto const& outputShape = outputIds.getShape();
    SizeType const maxSeqLength = outputShape.d[0];
    SizeType const beamWidth = outputShape.d[2];

    auto const inputBatchSize = inputIds.getSize() / maxInputLength;
    TLLM_CHECK_WITH_INFO(std::size_t(batchSize) == inputBatchSize,
        common::fmtstr("Input ids batch size (%ld) does not match inputLengths size (%ld)", inputBatchSize,
            std::size_t(batchSize)));
    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[1],
        common::fmtstr(
            "Output ids batch size (%d) does not match inputLengths size (%d)", outputShape.d[1], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyInputToOutputTransposed<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(outputIds),
        bufferCast<SizeType const>(inputIds), bufferCast<SizeType const>(inputLengths), padId, batchSize, beamWidth,
        maxInputLength);
}

namespace
{
__global__ void copyPackedInputToOutputTransposed(SizeType* outputIds, SizeType const* inputIds,
    SizeType const* inputOffsets, SizeType const padId, SizeType const batchSize, SizeType const beamWidth,
    SizeType const maxInputLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const tokenBegin = inputOffsets[batchIdx];
        auto const tokenEnd = inputOffsets[batchIdx + 1];
        auto const inputLength = tokenEnd - tokenBegin;

        for (SizeType tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[tokenBegin + tokenIdx] : padId;
            for (SizeType beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(tokenIdx, batchIdx, beamIdx, batchSize, beamWidth);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyPackedInputToOutputTransposed(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputOffsets,
    SizeType const maxInputLength, SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const batchSize = static_cast<SizeType>(inputOffsets.getSize()) - 1;
    auto const& outputShape = outputIds.getShape();
    SizeType const maxSeqLength = outputShape.d[0];
    SizeType const beamWidth = outputShape.d[2];

    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[1],
        common::fmtstr(
            "Output ids batch size (%d) does not match inputOffsets batch size (%d)", outputShape.d[1], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyPackedInputToOutputTransposed<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(outputIds),
        bufferCast<SizeType const>(inputIds), bufferCast<SizeType const>(inputOffsets), padId, batchSize, beamWidth,
        maxInputLength);
}

namespace
{
__global__ void copyInputToOutput(SizeType* outputIds, SizeType const* inputIds, SizeType const* inputLengths,
    SizeType const padId, SizeType const batchSize, SizeType const beamWidth, SizeType const maxInputLength,
    SizeType const maxSeqLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[batchIdx * maxInputLength + tokenIdx] : padId;
            for (SizeType beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(batchIdx, beamIdx, tokenIdx, beamWidth, maxSeqLength);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyInputToOutput(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const& inputShape = inputIds.getShape();
    auto const& outputShape = outputIds.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 3, common::fmtstr("Output shape must have 3 dimensions, but has %d", outputShape.nbDims));

    auto const batchSize = static_cast<SizeType>(inputLengths.getSize());
    SizeType const maxInputLength = inputShape.d[inputShape.nbDims - 1];
    SizeType const beamWidth = outputShape.d[1];
    SizeType const maxSeqLength = outputShape.d[2];

    auto const inputBatchSize = inputIds.getSize() / maxInputLength;
    TLLM_CHECK_WITH_INFO(std::size_t(batchSize) == inputBatchSize,
        common::fmtstr("Input ids batch size (%ld) does not match inputLengths size (%ld)", inputBatchSize,
            std::size_t(batchSize)));
    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[0],
        common::fmtstr(
            "Output ids batch size (%d) does not match inputLengths size (%d)", outputShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyInputToOutput<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(outputIds),
        bufferCast<SizeType const>(inputIds), bufferCast<SizeType const>(inputLengths), padId, batchSize, beamWidth,
        maxInputLength, maxSeqLength);
}

namespace
{
__global__ void copyPackedInputToOutput(SizeType* outputIds, SizeType const* inputIds, SizeType const* inputOffsets,
    SizeType const padId, SizeType const batchSize, SizeType const beamWidth, SizeType const maxInputLength,
    SizeType const maxSeqLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const tokenBegin = inputOffsets[batchIdx];
        auto const tokenEnd = inputOffsets[batchIdx + 1];
        auto const inputLength = tokenEnd - tokenBegin;

        for (SizeType tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[tokenBegin + tokenIdx] : padId;
            for (SizeType beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(batchIdx, beamIdx, tokenIdx, beamWidth, maxSeqLength);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyPackedInputToOutput(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputOffsets,
    SizeType const maxInputLength, SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const& outputShape = outputIds.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 3, common::fmtstr("Output shape must have 3 dimensions, but has %d", outputShape.nbDims));

    auto const batchSize = static_cast<SizeType>(inputOffsets.getSize()) - 1;
    SizeType const beamWidth = outputShape.d[1];
    SizeType const maxSeqLength = outputShape.d[2];

    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[0],
        common::fmtstr(
            "Output ids batch size (%d) does not match inputOffsets batch size (%d)", outputShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyPackedInputToOutput<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(outputIds),
        bufferCast<SizeType const>(inputIds), bufferCast<SizeType const>(inputOffsets), padId, batchSize, beamWidth,
        maxInputLength, maxSeqLength);
}

namespace
{
template <typename T>
__global__ void scatterTensor(T* output, T const* input, std::uint32_t const batchSize,
    std::uint32_t const inputRowSize, std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = batchIdx * inputRowSize + columnIdx;
            auto const value = input[inputIdx];
            std::size_t constexpr beamIdx{0};
            auto const outputIdx = (batchIdx * beamWidth + beamIdx) * outputRowSize + columnIdx;
            output[outputIdx] = value;
        }
    }
}

template <typename T>
__global__ void tileTensor(T* output, T const* input, std::uint32_t const batchSize, std::size_t const inputRowSize,
    std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = batchIdx * inputRowSize + columnIdx;
            auto const value = input[inputIdx];
            for (std::size_t beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = (batchIdx * beamWidth + beamIdx) * outputRowSize + columnIdx;
                output[outputIdx] = value;
            }
        }
    }
}

template <typename T>
__global__ void tileTensorInPlace(
    T* inputOutput, std::uint32_t const batchSize, std::size_t const inputOutputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputOutputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = (batchIdx * beamWidth + 0) * inputOutputRowSize + columnIdx;
            auto const value = inputOutput[inputIdx];
            for (std::size_t beamIdx = 1; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = (batchIdx * beamWidth + beamIdx) * inputOutputRowSize + columnIdx;
                inputOutput[outputIdx] = value;
            }
        }
    }
}

} // namespace

template <typename T>
void invokeScatterTensor(ITensor& output, ITensor const& input, SizeType beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    scatterTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input),
        nbInputRows, inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

void scatterTensor(ITensor& output, ITensor const& input, SizeType beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeScatterTensor<SizeType>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeScatterTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeScatterTensor<half>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeScatterTensor<int8_t>(output, input, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeScatterTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
#endif
    default: TLLM_CHECK_WITH_INFO(false, "data type not supported");
    }
}

template <typename T>
void invokeTileTensor(ITensor& output, ITensor const& input, SizeType const beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    tileTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input), nbInputRows,
        inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

void tileTensor(ITensor& output, ITensor const& input, SizeType beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeTileTensor<SizeType>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeTileTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeTileTensor<half>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeTileTensor<int8_t>(output, input, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeTileTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
#endif
    default: TLLM_CHECK_WITH_INFO(false, "data type not supported");
    }
}

template <typename T>
void invokeTileTensorInPlace(ITensor& inputOutput, SizeType const beamWidth, CudaStream const& stream)
{
    auto const& inputOutputShape = inputOutput.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(inputOutputShape.d[0]);
    auto const nbInputRows = nbOutputRows / static_cast<std::uint32_t>(beamWidth);
    auto const inputOutputRowSize = inputOutput.getSize() / static_cast<std::size_t>(nbOutputRows);

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputOutputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    tileTensorInPlace<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<T>(inputOutput), nbInputRows, inputOutputRowSize, static_cast<std::uint32_t>(beamWidth));
}

void tileTensorInplace(ITensor& tensor, SizeType beamWidth, CudaStream const& stream)
{
    switch (tensor.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeTileTensorInPlace<SizeType>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeTileTensorInPlace<float>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeTileTensorInPlace<half>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeTileTensorInPlace<int8_t>(tensor, beamWidth, stream); break;
#ifdef ENABLE_FP8    
    case nvinfer1::DataType::kFP8: invokeTileTensorInPlace<__hip_fp8_e4m3_fnuz>(tensor, beamWidth, stream); break;
#endif
    default: TLLM_CHECK_WITH_INFO(false, "data type not supported");
    }
}

// In the following kernel, we launch a grid with batchSize blocks of threads. Each thread block
// copies the logits from the "logits" tensor to the "lastTokenLogits" tensor for the last token
// of each sequence.
//
// TODO: Enable vector copies for higher BW utilization.

template <typename T>
__global__ void gatherLastTokenLogitsKernel(T* lastTokenLogits, T const* logits, int const* lastTokenIds,
    int maxInputLength, int beamWidth, int vocabSizePadded)
{
    // This sequence.
    int seqIdx = blockIdx.x;
    // Find the index of the last token in that sequence.
    // Since lastTokenIds is the accumulated length instead of real ids, so we need to minus 1.
    // For length [11, 23], we hope to get the results of id 10 and 22, in fact.
    int lastTokenIdx = lastTokenIds[seqIdx] - 1;

    // The output pointer.
    T* lastTokenLogitsPtr = &lastTokenLogits[seqIdx * beamWidth * vocabSizePadded];
    // The input pointer.
    T const* logitsPtr = &logits[lastTokenIdx * vocabSizePadded];

    // The threads in the block collaborate to copy the logits.
    for (int idx = threadIdx.x; idx < vocabSizePadded; idx += blockDim.x)
    {
        T value = logitsPtr[idx];
        for (int beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
        {
            lastTokenLogitsPtr[beamIdx * vocabSizePadded + idx] = value;
        }
    }
}

template <typename T>
void invokeGatherLastTokenLogits(
    ITensor& output, ITensor const& input, ITensor const& lastTokenIds, CudaStream const& stream)
{
    auto const& outputShape = output.getShape();
    auto const batchSize = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const beamWidth = static_cast<std::uint32_t>(outputShape.d[1]);
    auto const vocabSizePadded = static_cast<std::uint32_t>(outputShape.d[2]);

    auto const& inputShape = input.getShape();
    auto const maxInputLength = static_cast<std::uint32_t>(inputShape.d[1]);

    TLLM_CHECK_WITH_INFO(inputShape.d[0] == batchSize, "Invalid input shape: dim[0]");
    TLLM_CHECK_WITH_INFO(inputShape.d[2] == vocabSizePadded, "Invalid input shape: dim[2]");

    dim3 const blockSize{256, 1};
    dim3 const gridSize{static_cast<std::uint32_t>(batchSize), 1};
    gatherLastTokenLogitsKernel<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T>(input),
        bufferCast<int32_t>(lastTokenIds), static_cast<std::uint32_t>(maxInputLength),
        static_cast<std::uint32_t>(beamWidth), vocabSizePadded);
}

void gatherLastTokenLogits(ITensor& output, ITensor const& input, ITensor const& lastTokenIds, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kFLOAT: invokeGatherLastTokenLogits<float>(output, input, lastTokenIds, stream); break;
    case nvinfer1::DataType::kHALF: invokeGatherLastTokenLogits<half>(output, input, lastTokenIds, stream); break;
#if defined(NV_TENSORRT_MAJOR) && NV_TENSORRT_MAJOR >= 9
    case nvinfer1::DataType::kBF16:
        invokeGatherLastTokenLogits<__hip_bfloat16>(output, input, lastTokenIds, stream);
        break;
#endif
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8:
        invokeGatherLastTokenLogits<__hip_fp8_e4m3_fnuz>(output, input, lastTokenIds, stream);
        break;
#endif
    default: TLLM_CHECK_WITH_INFO(false, "data type not supported");
    }
}

} // namespace tensorrt_llm::runtime::kernels
